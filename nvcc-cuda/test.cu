#include "hip/hip_runtime.h"

CUDA_PROGRAM((

__global__ void add(int n, float* x, float* y) {
	for (int i = threadIdx.x; i < n; i += blockDim.x) {
		y[i] += x[i];
		
	}
	
}

int main(void) {
	int ARRAY_SIZE = 1 << 24;
	
	float* x;
	float* y;
	
	hipMallocManaged(&x, ARRAY_SIZE * sizeof(float));
	hipMallocManaged(&y, ARRAY_SIZE * sizeof(float));
	
	int i;
	for (i = 0; i < ARRAY_SIZE; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
		
	}
	
	add<<<1, 1024>>>(ARRAY_SIZE, x, y);
	hipDeviceSynchronize();
	
	hipFree(x);
	hipFree(y);
	
	return 0;
	
}

))
